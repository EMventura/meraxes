#include "hip/hip_runtime.h"
#include "meraxes.h"
#include "meraxes_gpu.h"
#include "utils.h"
#include <fftw3.h>
#include <fftw3-mpi.h>
#include <math.h>
#include <assert.h>
#include <signal.h>
#include <limits.h>

#include <hdf5.h>
#include <hdf5_hl.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

__global__
void complex_vector_times_scalar(Complex *vector,double scalar,int n){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n){
        vector[i].x*=scalar;
        vector[i].y*=scalar;
    }
}

__global__
void sanity_check_aliasing(Complex *grid,int n,float val){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n)
          ((float *)grid)[i] = fmaxf(((float *)grid)[i], val);
}

__device__ void inline index2indices_FFT_k(const int dim,int index,int *i_k);
__device__ void inline index2indices_FFT_k(const int dim,int index,int *i_k){ // should match mode=INDEX_COMPLEX_HERM in grid_index ... check this!
  int i_d,j_d;
  int remainder;
  for(j_d=2,remainder=index;j_d>=0;j_d--){
    i_d=j_d;
    i_k[i_d]  =remainder%dim;
    remainder-=i_k[i_d];
    remainder/=dim;
  }
}

__device__ float k_mag_of_index(const int dim,int index);
__device__ float k_mag_of_index(const int dim,int index){
    int idxs[3];
    index2indices_FFT_k(dim,index,idxs);
    float k_mag = 0.f;
    return(k_mag);
}

__global__
void filter_gpu(Complex *grid,int dim,int n,float R){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n){
        float kR = R*k_mag_of_index(dim,i);
        float scalar =0.;
        int   support=false;
int filter_type = 0;
        switch(filter_type)
        {
          case 0:   // Real space top-hat
            scalar  = (3.0 * (sinf(kR) / powf(kR, 3) - cosf(kR) / powf(kR, 2)));
            support = (kR>1e-4);
            break;

          case 1:                  // k-space top hat
            kR     *= 0.413566994; // Equates integrated volume to the real space top-hat (9pi/2)^(-1/3)
            scalar  = 0.f;
            support = (kR>1);
            break;

          case 2:        // Gaussian
            kR     *= 0.643; // Equates integrated volume to the real space top-hat
            scalar  = powf(M_E, -kR * kR / 2.0);
            support = true;
            break;

          // Implement this check before the kernel!!!!!!!!!!
          //default:
          //  if (i==0)
          //  {
          //    mlog_error("ReionFilterType.c: Warning, ReionFilterType type %d is undefined!", filter_type);
          //    ABORT(EXIT_FAILURE);
          //  }
          //  break;
        }
        if(support){
            grid[i].x*=scalar;
            grid[i].y*=scalar;
        }
    }
}


// Presently, this is just a copy of what's in Meraxes
void _find_HII_bubbles_gpu(
    // input
    double redshift,
    MPI_Comm mpi_comm,
    int mpi_rank,
    double box_size,
    int ReionGridDim,
    int local_nix,
    int flag_ReionUVBFlag,
    double ReionEfficiency,
    double ReionNionPhotPerBary,
    double UnitLength_in_cm,
    double UnitMass_in_g,
    double UnitTime_in_s,
    double ReionRBubbleMax,
    double ReionRBubbleMin,
    double ReionDeltaRFactor,
    double ReionGammaHaloBias,
    double ReionAlphaUV,
    double ReionEscapeFrac,

    bool validation_output,

    // preallocated 1D grids (local_nix * ReionGridDim * ReionGridDim)
    float *J_21,  // real
    float *r_bubble, // real

    // input grids
    float *deltax,  // real & padded
    float *stars,  // real & padded
    float *sfr,  // real & padded

    // preallocated
    Complex *deltax_filtered_in,  // complex
    Complex *stars_filtered_in,  // complex
    Complex *sfr_filtered_in,  // complex

    // length = mpi.size
    ptrdiff_t *slabs_n_complex,
    ptrdiff_t *slabs_ix_start,

    // output - preallocated real grids (local_nix * ReionGridDim * ReionGridDim)
    float *xH, // real
    float *z_at_ionization,
    float *J_21_at_ionization,

    // output - single values
    double *volume_weighted_global_xH,
    double *mass_weighted_global_xH
    )
{
  const double pixel_volume         = pow(box_size / (double)ReionGridDim, 3); // (Mpc/h)^3
  const double total_n_cells        = pow((double)ReionGridDim, 3);
  const double inv_total_n_cells    = 1.f/total_n_cells;
  const int    slab_n_real          = local_nix * ReionGridDim * ReionGridDim;
  const int    slab_n_complex       = (int)(slabs_n_complex[mpi_rank]);
  double       cell_length_factor   = L_FACTOR;


  if (validation_output)
  {
    // prepare output file
    char fname[STRLEN];
    sprintf(fname, "validation_input-core%03d-z%.2f.h5", mpi_rank, redshift);
    hid_t file_id = H5Fcreate(fname, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);

    // write all of the input values
    H5LTset_attribute_double(file_id, "/", "redshift", &redshift, 1);
    H5LTset_attribute_int(file_id, "/", "mpi_rank", &mpi_rank, 1);
    H5LTset_attribute_double(file_id, "/", "box_size", &box_size, 1);
    H5LTset_attribute_int(file_id, "/", "ReionGridDim", &ReionGridDim, 1);
    H5LTset_attribute_int(file_id, "/", "local_nix", &local_nix, 1);
    H5LTset_attribute_int(file_id, "/", "flag_ReionUVBFlag", &flag_ReionUVBFlag, 1);
    H5LTset_attribute_double(file_id, "/", "ReionEfficiency", &ReionEfficiency, 1);
    H5LTset_attribute_double(file_id, "/", "ReionNionPhotPerBary", &ReionNionPhotPerBary, 1);
    H5LTset_attribute_double(file_id, "/", "UnitLength_in_cm", &UnitLength_in_cm, 1);
    H5LTset_attribute_double(file_id, "/", "UnitMass_in_g", &UnitMass_in_g, 1);
    H5LTset_attribute_double(file_id, "/", "UnitTime_in_s", &UnitTime_in_s, 1);
    H5LTset_attribute_double(file_id, "/", "ReionRBubbleMax", &ReionRBubbleMax, 1);
    H5LTset_attribute_double(file_id, "/", "ReionRBubbleMin", &ReionRBubbleMin, 1);
    H5LTset_attribute_double(file_id, "/", "ReionDeltaRFactor", &ReionDeltaRFactor, 1);
    H5LTset_attribute_double(file_id, "/", "ReionGammaHaloBias", &ReionGammaHaloBias, 1);
    H5LTset_attribute_double(file_id, "/", "ReionAlphaUV", &ReionAlphaUV, 1);
    H5LTset_attribute_double(file_id, "/", "ReionEscapeFrac", &ReionEscapeFrac, 1);

    H5LTmake_dataset_float(file_id, "deltax", 1, (hsize_t []){slab_n_complex*2}, deltax);
    H5LTmake_dataset_float(file_id, "stars", 1, (hsize_t []){slab_n_complex*2}, stars);
    H5LTmake_dataset_float(file_id, "sfr", 1, (hsize_t []){slab_n_complex*2}, sfr);

    H5Fclose(file_id);
  }

  // This parameter choice is sensitive to noise on the cell size, at least for the typical
  // cell sizes in RT simulations. It probably doesn't matter for larger cell sizes.
  if ((box_size / (double)ReionGridDim) < 1.0) // Fairly arbitrary length based on 2 runs Sobacchi did
    cell_length_factor = 1.0;

  // Init J_21
  int ii;
  if (flag_ReionUVBFlag)
    for(ii = 0; ii < slab_n_real; ii++)
      J_21[ii] = 0.0;

  // Init xH
  for(ii = 0; ii < slab_n_real; ii++)
    xH[ii] = 1.0;

  // Init r_bubble
  for(ii = 0; ii < slab_n_real; ii++)
    r_bubble[ii] = 0.0;

  // Forward fourier transform to obtain k-space fields
  // Initialize cuFFT
  hipfftHandle plan;
  hipfftPlan3d(&plan, ReionGridDim, ReionGridDim, ReionGridDim, HIPFFT_R2C);
  cufftSetCompatibilityMode(plan,CUFFT_COMPATIBILITY_FFTW_ALL);

  // Initialize arrays on the device
  hipfftComplex *deltax_unfiltered = (hipfftComplex *)deltax;
  hipfftComplex *stars_unfiltered  = (hipfftComplex *)stars;
  hipfftComplex *sfr_unfiltered    = (hipfftComplex *)sfr;
  hipfftComplex *deltax_filtered   = NULL;
  hipfftComplex *stars_filtered    = NULL;
  hipfftComplex *sfr_filtered      = NULL;
  hipMalloc((void**)&deltax_unfiltered,sizeof(hipfftComplex)*slab_n_complex);
  hipMalloc((void**)&stars_unfiltered, sizeof(hipfftComplex)*slab_n_complex);
  hipMalloc((void**)&sfr_unfiltered,   sizeof(hipfftComplex)*slab_n_complex);
  hipMalloc((void**)&deltax_filtered,  sizeof(hipfftComplex)*slab_n_complex);
  hipMalloc((void**)&stars_filtered,   sizeof(hipfftComplex)*slab_n_complex);
  hipMalloc((void**)&sfr_filtered,     sizeof(hipfftComplex)*slab_n_complex);
  hipMemcpy(deltax,deltax_unfiltered,  sizeof(hipfftComplex)*slab_n_complex,hipMemcpyHostToDevice);
  hipMemcpy(stars, stars_unfiltered,   sizeof(hipfftComplex)*slab_n_complex,hipMemcpyHostToDevice);
  hipMemcpy(sfr,   sfr_unfiltered,     sizeof(hipfftComplex)*slab_n_complex,hipMemcpyHostToDevice);

  // Perform FFTs
  if (hipfftExecR2C(plan,(hipfftReal *)deltax_unfiltered,deltax_unfiltered) != HIPFFT_SUCCESS ) {
    fprintf(stderr, "Cuda error 1.\n");
    return ;
  }
  if (hipfftExecR2C(plan,(hipfftReal *)stars_unfiltered,stars_unfiltered) != HIPFFT_SUCCESS ) {
    fprintf(stderr, "Cuda error 2.\n");
    return ;
  }
  if (hipfftExecR2C(plan,(hipfftReal *)sfr_unfiltered,sfr_unfiltered) != HIPFFT_SUCCESS ) {
    fprintf(stderr, "Cuda error 3.\n");
    return ;
  }

  // Make sure that the device has synchronized
  if(hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error 4.\n");
    return;
  }

  // Clean-up the device
  hipfftDestroy(plan);

  if (validation_output)
  {
    // prepare output file
    char fname[STRLEN];
    sprintf(fname, "validation_output-core%03d-z%.2f.h5", mpi_rank, redshift);
    hid_t file_id = H5Fcreate(fname, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);

    hid_t group = H5Gcreate(file_id, "kspace", H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT);

    H5LTmake_dataset_float(group, "deltax", 1, (hsize_t []){slab_n_complex * 2}, deltax);
    H5LTmake_dataset_float(group, "stars", 1, (hsize_t []){slab_n_complex * 2}, stars);
    H5LTmake_dataset_float(group, "sfr", 1, (hsize_t []){slab_n_complex * 2}, sfr);

    H5Gclose(group);
    H5Fclose(file_id);
  }

  // Remember to add the factor of VOLUME/TOT_NUM_PIXELS when converting from real space to k-space
  // Note: we will leave off factor of VOLUME, in anticipation of the inverse FFT below
  int threads = 256;  
  int grid    = (slab_n_complex+255)/256;
  complex_vector_times_scalar<<<grid, threads>>>(deltax_unfiltered,inv_total_n_cells,slab_n_complex);
  complex_vector_times_scalar<<<grid, threads>>>(stars_unfiltered, inv_total_n_cells,slab_n_complex);
  complex_vector_times_scalar<<<grid, threads>>>(sfr_unfiltered,   inv_total_n_cells,slab_n_complex);

  // Loop through filter radii
  double R                     = fmin(ReionRBubbleMax, L_FACTOR * box_size); // Mpc/h

  bool  flag_last_filter_step = false;

  while(!flag_last_filter_step)
  {
    // check to see if this is our last filtering step
    if( ((R / ReionDeltaRFactor) <= (cell_length_factor * box_size / (double)ReionGridDim))
        || ((R / ReionDeltaRFactor) <= ReionRBubbleMin) )
    {
      flag_last_filter_step = true;
      R                     = cell_length_factor * box_size / (double)ReionGridDim;
    }

    mlog(".", MLOG_CONT);

    // copy the k-space grids
    hipMemcpy(deltax_filtered,deltax_unfiltered,sizeof(Complex) * slab_n_complex,hipMemcpyDeviceToDevice);
    hipMemcpy(stars_filtered, stars_unfiltered, sizeof(Complex) * slab_n_complex,hipMemcpyDeviceToDevice);
    hipMemcpy(sfr_filtered,   sfr_unfiltered,   sizeof(Complex) * slab_n_complex,hipMemcpyDeviceToDevice);
    if(!flag_last_filter_step){
       filter_gpu<<<grid,threads>>>(deltax_filtered,ReionGridDim,slab_n_complex,(float)R);
       filter_gpu<<<grid,threads>>>(stars_filtered, ReionGridDim,slab_n_complex,(float)R);
       filter_gpu<<<grid,threads>>>(sfr_filtered,   ReionGridDim,slab_n_complex,(float)R);
    }

    // inverse fourier transform back to real space
    // Initialize cuFFT
    hipfftPlan3d(&plan, ReionGridDim, ReionGridDim, ReionGridDim, HIPFFT_C2R);
    cufftSetCompatibilityMode(plan,CUFFT_COMPATIBILITY_FFTW_ALL);
    
    // Perform FFTs
    if (hipfftExecC2R(plan,(hipfftComplex *)deltax_filtered, (hipfftReal *)deltax_filtered) != HIPFFT_SUCCESS ) {
      fprintf(stderr, "Cuda error 101.\n");
      return ;
    }
    if (hipfftExecC2R(plan,(hipfftComplex *)stars_filtered, (hipfftReal *)stars_filtered) != HIPFFT_SUCCESS ) {
      fprintf(stderr, "Cuda error 102.\n");
      return ;
    }
    if (hipfftExecC2R(plan,(hipfftComplex *)sfr_filtered, (hipfftReal *)sfr_filtered) != HIPFFT_SUCCESS ) {
      fprintf(stderr, "Cuda error 103.\n");
      return ;
    }

    // Clean-up device
    hipfftDestroy(plan);

    // Perform sanity checks to account for aliasing effects
    sanity_check_aliasing<<<grid,threads>>>(deltax_filtered,slab_n_complex,-1.f + REL_TOL);
    sanity_check_aliasing<<<grid,threads>>>(stars_filtered, slab_n_complex,0.);
    sanity_check_aliasing<<<grid,threads>>>(sfr_filtered,   slab_n_complex,0.);

    /*
     * Main loop through the box...
     */

    const double J_21_aux_constant = (1.0 + redshift) * (1.0 + redshift) / (4.0 * M_PI)
      * ReionAlphaUV * PLANCK
      * 1e21 * ReionEscapeFrac
      * R *UnitLength_in_cm * ReionNionPhotPerBary / PROTONMASS
      * UnitMass_in_g / pow(UnitLength_in_cm, 3) / UnitTime_in_s;

#ifdef __NVCC__
#else
    const double inv_pixel_volume = 1.f/pixel_volume;
    for (ix = 0; ix < local_nix; ix++)
      for (iy = 0; iy < ReionGridDim; iy++)
        for (iz = 0; iz < ReionGridDim; iz++)
        {
          const int i_real   = grid_index(ix, iy, iz, ReionGridDim, INDEX_REAL);
          const int i_padded = grid_index(ix, iy, iz, ReionGridDim, INDEX_PADDED);

          double density_over_mean = 1.0 + (double)((float *)deltax_filtered)[i_padded];

          double f_coll_stars      =  (double)((float *)stars_filtered)[i_padded] / (RtoM(R) * density_over_mean)
                               * (4.0 / 3.0) * M_PI * pow(R,3.0)  * inv_pixel_volume;

          double sfr_density       = (double)((float *)sfr_filtered)[i_padded] * inv_pixel_volume; // In internal units

          float J_21_aux;
          if (flag_ReionUVBFlag)
            J_21_aux = (float)(sfr_density * J_21_aux_constant);

          // Check if ionised!
          if (f_coll_stars > 1.0 / ReionEfficiency)   // IONISED!!!!
          {
            // If it is the first crossing of the ionisation barrier for this cell (largest R), let's record J_21
            if (xH[i_real] > REL_TOL)
              if(flag_ReionUVBFlag)
                J_21[i_real] = J_21_aux;

            // Mark as ionised
            xH[i_real]       = 0;

            // Record radius
            r_bubble[i_real] = (float)R;
          }
          // Check if this is the last filtering step.
          // If so, assign partial ionisations to those cells which aren't fully ionised
          else if (flag_last_filter_step && (xH[i_real] > REL_TOL))
            xH[i_real] = (float)(1.0 - f_coll_stars * ReionEfficiency);

          // Check if new ionisation
          float *z_in = z_at_ionization;
          if ( (xH[i_real] < REL_TOL) && (z_in[i_real] < 0) )   // New ionisation!
          {
            z_in[i_real] = (float)redshift;
            if (flag_ReionUVBFlag)
              J_21_at_ionization[i_real] = J_21_aux * (float)ReionGammaHaloBias;
          }
        }
    // iz
#endif

    R /= ReionDeltaRFactor;
  }


  // Find the volume and mass weighted neutral fractions
  // TODO: The deltax grid will have rounding errors from forward and reverse
  //       FFT. Should cache deltax slabs prior to ffts and reuse here.
  *volume_weighted_global_xH = 0.0;
  *mass_weighted_global_xH   = 0.0;
  double mass_weight         = 0.0;

#ifdef __NVCC__
#else
  for (ix = 0; ix < local_nix; ix++)
    for (iy = 0; iy < ReionGridDim; iy++)
      for (iz = 0; iz < ReionGridDim; iz++)
      {
        const int i_real   = grid_index(ix, iy, iz, ReionGridDim, INDEX_REAL);
        const int i_padded = grid_index(ix, iy, iz, ReionGridDim, INDEX_PADDED);
        double density_over_mean    = 1.0 + (double)((float *)deltax_filtered)[i_padded];
        *volume_weighted_global_xH += (double)xH[i_real];
        *mass_weighted_global_xH   += (double)(xH[i_real]) * density_over_mean;
        mass_weight                += density_over_mean;
      }
#endif

  MPI_Allreduce(MPI_IN_PLACE, &volume_weighted_global_xH, 1, MPI_DOUBLE, MPI_SUM, mpi_comm);
  MPI_Allreduce(MPI_IN_PLACE, &mass_weighted_global_xH, 1, MPI_DOUBLE, MPI_SUM, mpi_comm);
  MPI_Allreduce(MPI_IN_PLACE, &mass_weight, 1, MPI_DOUBLE, MPI_SUM, mpi_comm);

  *volume_weighted_global_xH                        *= inv_total_n_cells;
  *mass_weighted_global_xH                          /= mass_weight;

  // Clean-up
  hipFree(deltax_unfiltered);
  hipFree(stars_unfiltered);
  hipFree(sfr_unfiltered);
  hipFree(deltax_filtered);
  hipFree(stars_filtered);
  hipFree(sfr_filtered);

  if (validation_output)
  {
    // prepare output file
    char fname[STRLEN];
    sprintf(fname, "validation_output-core%03d-z%.2f.h5", mpi_rank, redshift);
    hid_t file_id = H5Fopen(fname, H5F_ACC_RDWR, H5P_DEFAULT);

    H5LTmake_dataset_float(file_id, "xH", 1, (hsize_t []){slab_n_real}, xH);
    H5LTmake_dataset_float(file_id, "z_at_ionization", 1, (hsize_t []){slab_n_real}, z_at_ionization);
    H5LTmake_dataset_float(file_id, "J_21_at_ionization", 1, (hsize_t []){slab_n_real}, J_21_at_ionization);

    H5LTset_attribute_double(file_id, "/", "volume_weighted_global_xH", volume_weighted_global_xH, 1);
    H5LTset_attribute_double(file_id, "/", "mass_weighted_global_xH", mass_weighted_global_xH, 1);

    H5Fclose(file_id);
  }
}

void find_HII_bubbles_driver(
    double redshift,
    void  (*_find_HII_bubbles_passed)(
        // input
        double redshift,
        MPI_Comm mpi_comm,
        int mpi_rank,
        double box_size,
        int ReionGridDim,
        int local_nix,
        int flag_ReionUVBFlag,
        double ReionEfficiency,
        double ReionNionPhotPerBary,
        double UnitLength_in_cm,
        double UnitMass_in_g,
        double UnitTime_in_s,
        double ReionRBubbleMax,
        double ReionRBubbleMin,
        double ReionDeltaRFactor,
        double ReionGammaHaloBias,
        double ReionAlphaUV,
        double ReionEscapeFrac,
    
        bool validation_output,
    
        // preallocated 1D grids (local_nix * ReionGridDim * ReionGridDim)
        float *J_21,  // real
        float *r_bubble, // real
    
        // input grids
        float *deltax,  // real & padded
        float *stars,  // real & padded
        float *sfr,  // real & padded
    
        // preallocated
        Complex *deltax_filtered_in,  // complex
        Complex *stars_filtered_in,  // complex
        Complex *sfr_filtered_in,  // complex
    
        // length = mpi.size
        ptrdiff_t *slabs_n_complex,
        ptrdiff_t *slabs_ix_start,
    
        // output - preallocated real grids (local_nix * ReionGridDim * ReionGridDim)
        float *xH, // real
        float *z_at_ionization,
        float *J_21_at_ionization,
    
        // output - single values
        double *volume_weighted_global_xH,
        double *mass_weighted_global_xH
        ),
    const char *reference_directory,
    timer_info *timer)
{
    int mpi_rank=run_globals.mpi_rank;
    int n_rank  =run_globals.mpi_size;
    if(n_rank!=1){
        if(mpi_rank==0) fprintf(stderr,"n_rank=%d but only n_rank==1 supported at this point.\n",n_rank);
        exit(1);
    }
    
    // Open inputs file
    char fname[STRLEN];
    sprintf(fname, "%s/validation_input-core%03d-z%.2f.h5",reference_directory,mpi_rank, redshift);
    hid_t file_id = H5Fopen(fname, H5F_ACC_RDONLY, H5P_DEFAULT);
    
    // Read input attributes
    double box_size;
    int    ReionGridDim;
    int    local_nix;
    int    flag_ReionUVBFlag;
    double ReionEfficiency;
    double ReionNionPhotPerBary;
    double UnitLength_in_cm;
    double UnitMass_in_g;
    double UnitTime_in_s;
    double ReionRBubbleMax;
    double ReionRBubbleMin;
    double ReionDeltaRFactor;
    double ReionGammaHaloBias;
    double ReionAlphaUV;
    double ReionEscapeFrac;
    H5LTget_attribute_double(file_id, "/", "redshift", &redshift);
    H5LTget_attribute_int   (file_id, "/", "mpi_rank", &mpi_rank);
    H5LTget_attribute_double(file_id, "/", "box_size", &box_size);
    H5LTget_attribute_int   (file_id, "/", "ReionGridDim", &ReionGridDim);
    H5LTget_attribute_int   (file_id, "/", "local_nix", &local_nix);
    H5LTget_attribute_int   (file_id, "/", "flag_ReionUVBFlag", &flag_ReionUVBFlag);
    H5LTget_attribute_double(file_id, "/", "ReionEfficiency", &ReionEfficiency);
    H5LTget_attribute_double(file_id, "/", "ReionNionPhotPerBary", &ReionNionPhotPerBary);
    H5LTget_attribute_double(file_id, "/", "UnitLength_in_cm", &UnitLength_in_cm);
    H5LTget_attribute_double(file_id, "/", "UnitMass_in_g", &UnitMass_in_g);
    H5LTget_attribute_double(file_id, "/", "UnitTime_in_s", &UnitTime_in_s);
    H5LTget_attribute_double(file_id, "/", "ReionRBubbleMax", &ReionRBubbleMax);
    H5LTget_attribute_double(file_id, "/", "ReionRBubbleMin", &ReionRBubbleMin);
    H5LTget_attribute_double(file_id, "/", "ReionDeltaRFactor", &ReionDeltaRFactor);
    H5LTget_attribute_double(file_id, "/", "ReionGammaHaloBias", &ReionGammaHaloBias);
    H5LTget_attribute_double(file_id, "/", "ReionAlphaUV", &ReionAlphaUV);
    H5LTget_attribute_double(file_id, "/", "ReionEscapeFrac", &ReionEscapeFrac);

    // Initialize fftw here (we need to do this because we need 'slab_n_complex' to set the size of the input datasets
    ptrdiff_t local_nix_2;
    ptrdiff_t local_ix_start;
    int       local_n_complex = fftwf_mpi_local_size_3d(ReionGridDim, ReionGridDim, ReionGridDim / 2 + 1, run_globals.mpi_comm, &local_nix_2, &local_ix_start);
    if(local_nix!=local_nix_2){
        printf("Error: local_nix!=local_nix_2 (ie. %d!=%d)\n",local_nix,local_nix_2);
        exit(1);
    }
    int       slab_n_complex    =  local_n_complex;
    int       slab_n_real       =  local_nix*ReionGridDim*ReionGridDim;
    ptrdiff_t slabs_n_complex[] = {slab_n_complex}; // works for 1 core only
    ptrdiff_t slabs_ix_start[]  = {0};              // works for 1 core only

    // Read input datasets
    float *deltax = (float *)malloc(sizeof(float)*(slab_n_complex * 2));
    float *stars  = (float *)malloc(sizeof(float)*(slab_n_complex * 2));
    float *sfr    = (float *)malloc(sizeof(float)*(slab_n_complex * 2));
    H5LTread_dataset_float(file_id, "deltax", deltax);
    H5LTread_dataset_float(file_id, "sfr",  sfr);
    H5LTread_dataset_float(file_id, "stars", stars);
    H5Fclose(file_id);

    // Initialize outputs    
    float   *J_21                     =(float   *)malloc(sizeof(float)  *slab_n_real);
    float   *r_bubble                 =(float   *)malloc(sizeof(float)  *slab_n_real); 
    Complex *deltax_filtered          =(Complex *)malloc(sizeof(Complex)*slab_n_complex);
    Complex *stars_filtered           =(Complex *)malloc(sizeof(Complex)*slab_n_complex);  
    Complex *sfr_filtered             =(Complex *)malloc(sizeof(float)  *slab_n_complex);
    float   *xH                       =(float   *)malloc(sizeof(float)  *slab_n_real);
    float   *z_at_ionization          =(float   *)calloc(slab_n_real,sizeof(float));
    float   *J_21_at_ionization       =(float   *)calloc(slab_n_real,sizeof(float));
    double   volume_weighted_global_xH;
    double   mass_weighted_global_xH;
    
    // Call the version of find_HII_bubbles we've been passed
    timer_start(timer);
    _find_HII_bubbles_passed(
        redshift, //
        run_globals.mpi_comm,
        mpi_rank,
        box_size, //
        ReionGridDim, //
        local_nix, //
        flag_ReionUVBFlag, //
        ReionEfficiency, //
        ReionNionPhotPerBary, //
        UnitLength_in_cm, //
        UnitMass_in_g, //
        UnitTime_in_s, //
        ReionRBubbleMax, //
        ReionRBubbleMin, //
        ReionDeltaRFactor, //
        ReionGammaHaloBias, //
        ReionAlphaUV, //
        ReionEscapeFrac, //
        true, //
        J_21,  
        r_bubble, 
        deltax, //
        stars, //
        sfr, //
        deltax_filtered,  
        stars_filtered,  
        sfr_filtered,  
        slabs_n_complex,
        slabs_ix_start,
        xH, 
        z_at_ionization,
        J_21_at_ionization,
        &volume_weighted_global_xH,
        &mass_weighted_global_xH);
    timer_stop(timer);
    
    // Clean-up
    free(J_21);
    free(r_bubble);
    free(deltax);
    free(stars);
    free(sfr);
    free(deltax_filtered);
    free(stars_filtered);
    free(sfr_filtered);
    free(xH);
    free(z_at_ionization);
    free(J_21_at_ionization);
}

